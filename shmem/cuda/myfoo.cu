
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <time.h>

//#include <common.h>
#define M 10
#define NR_BLOCK 1024

__global__ void compute(const float * a, float * b)
{
    int i = blockIdx.x;
    int j;    

    for (j = 0; j < M; j++) {
        if ((i + j * NR_BLOCK) > 0 && (i + j * NR_BLOCK) < M) {
            b[i + j * NR_BLOCK] = 0.2 * (a[M+((i+j*NR_BLOCK)-1)] + a[M+(i+j*NR_BLOCK)] + a[M+((i+j*NR_BLOCK)+1)] + a[(i+j*NR_BLOCK)] + a[2*M+(i+j*NR_BLOCK)]);
        }
    } 
}

struct params {
    float ** a;
    float ** b;
    float * c;
    float * d;
    float * c_a;
    float * c_b;
    int up, down, j;
    int stop;
    int num_pes;
    int mype;
};
typedef struct params params_t;

void foo(params_t * param)
{
    int j = param->j;
    int up = param->up;
    int down = param->down;
    int num_pes = param->num_pes;
    int mype = param->mype;
    if ((mype % 2) == 1) {
        hipSetDevice(1);
    } else {
        hipSetDevice(0);
    }
    // above
    if (up != -1 && j == 0) {
         hipMemcpy(param->c_a, 
                   param->c, 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    } else {
        hipMemcpy(param->c_a, 
                   param->a[j - 1], 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    } 
    // middle
    hipMemcpy(&(param->c_a[M]), 
               param->a[j], 
               M * sizeof(float), 
               hipMemcpyHostToDevice);

    // below
    if (down != -1 && j == param->stop - 1) {
        hipMemcpy(&(param->c_a[2 * M]), 
                   param->d, 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    } else {
        hipMemcpy(&(param->c_a[2 * M]), 
                   param->a[j + 1], 
                   M * sizeof(float), 
                   hipMemcpyHostToDevice);
    }
    
    hipMemcpy(param->c_b, 
               param->b[j], 
               M * sizeof(float), 
               hipMemcpyHostToDevice);

    compute<<<NR_BLOCK, 1>>>(param->c_a, param->c_b);
    hipMemcpy(param->b[j], param->c_b, M * sizeof(float), hipMemcpyDeviceToHost);
}

